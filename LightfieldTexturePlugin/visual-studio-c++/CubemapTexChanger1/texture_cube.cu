#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "main.h"

const int N = 7;
const int blocksize = 7;

__global__ void hello(char *a, int *b)
{
	a[threadIdx.x] += b[threadIdx.x];
}

extern "C" 
void cuda_test()
{
	DebugInUnity("This is testing to see if I can call code from texture_cube.cu");

	char a[N] = "Hello ";
	int b[N] = { 15, 10, 6, 0, -11, 1, 0 };

	char *ad;
	int *bd;
	const int csize = N * sizeof(char);
	const int isize = N * sizeof(int);

	printf("%s", a);
	std::string output;
	output += a;

	hipMalloc((void**)&ad, csize);
	hipMalloc((void**)&bd, isize);
	hipMemcpy(ad, a, csize, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, isize, hipMemcpyHostToDevice);

	dim3 dimBlock(blocksize, 1);
	dim3 dimGrid(1, 1);
	hello << <dimGrid, dimBlock >> >(ad, bd);
	hipMemcpy(a, ad, csize, hipMemcpyDeviceToHost);
	hipFree(ad);

	output += a;

	DebugInUnity(output);
}