#include "hip/hip_runtime.h"
/* ==========================================================================
   cudaHelloWorld.cu
   ==========================================================================

   Test hello world to make sure that CUDA is working

*/

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "textureChanger.h"
#include "utils.h"


// --------------------------------------------------------------------------
// Kernel
// --------------------------------------------------------------------------

__global__ void CudaKernelHelloWorld(char *a, int *b)
{
	a[threadIdx.x] += b[threadIdx.x];
}


// --------------------------------------------------------------------------
// Wrapper
// --------------------------------------------------------------------------

extern "C"
void CudaWrapperHelloWorld()
{
	DebugInUnity("This is testing to see if I can call code from texture_cube.cu");

	const int N = 7;
	const int blocksize = 7;

	char a[N] = "Hello ";
	int b[N] = { 15, 10, 6, 0, -11, 1, 0 };

	char *ad;
	int *bd;
	const int csize = N * sizeof(char);
	const int isize = N * sizeof(int);

	std::string output;
	output += a;

	hipMalloc((void**)&ad, csize);
	hipMalloc((void**)&bd, isize);
	hipMemcpy(ad, a, csize, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, isize, hipMemcpyHostToDevice);

	dim3 dimBlock(blocksize, 1);
	dim3 dimGrid(1, 1);
	CudaKernelHelloWorld << <dimGrid, dimBlock >> >(ad, bd);
	hipMemcpy(a, ad, csize, hipMemcpyDeviceToHost);
	hipFree(ad);

	output += a;

	DebugInUnity(output);
}